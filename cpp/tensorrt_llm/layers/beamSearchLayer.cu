#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2024, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "tensorrt_llm/common/memoryUtils.h"
#include "tensorrt_llm/kernels/beamSearchKernels.h"
#include "tensorrt_llm/layers/beamSearchLayer.h"
#include "tensorrt_llm/layers/fillBuffers.h"

using namespace tensorrt_llm::common;
using namespace tensorrt_llm::kernels;

namespace tensorrt_llm
{
namespace layers
{

template <typename T>
BeamSearchLayer<T>::BeamSearchLayer(runtime::SizeType vocab_size, runtime::SizeType vocab_size_padded,
    hipStream_t stream, std::shared_ptr<IAllocator> allocator)
    : BaseLayer(stream, std::move(allocator), nullptr)
    , mVocabSize(vocab_size)
    , mVocabSizePadded(vocab_size_padded)
{
    TLLM_LOG_TRACE(__PRETTY_FUNCTION__);
}

template <typename T>
BeamSearchLayer<T>::BeamSearchLayer(BeamSearchLayer<T> const& beam_search_layer)
    : BaseLayer(beam_search_layer)
    , mVocabSize(beam_search_layer.mVocabSize)
    , mVocabSizePadded(beam_search_layer.mVocabSizePadded)
    , mWorkspaceSize(beam_search_layer.mWorkspaceSize)
{
    TLLM_LOG_TRACE(__PRETTY_FUNCTION__);
}

template <typename T>
BeamSearchLayer<T>::~BeamSearchLayer()
{
    TLLM_LOG_TRACE(__PRETTY_FUNCTION__);
}

template <typename T>
void BeamSearchLayer<T>::setup(
    runtime::SizeType const batch_size, runtime::SizeType const beam_width, SetupParams const& setupParams)
{
    TLLM_LOG_TRACE("%s start", __PRETTY_FUNCTION__);
    TLLM_CHECK_WITH_INFO(
        beam_width <= nMaxBeamWidth, std::string("Beam width is larger than the maximum supported (64)."));

    mDiversityRateHost.resize(batch_size);
    mLengthPenaltyHost.resize(batch_size);
    mEarlyStoppingHost.resize(batch_size);
    allocateBuffer(batch_size, beam_width);

    FillBuffers const fillBuffers{batch_size, batch_size, mStream};
    fillBuffers(setupParams.beam_search_diversity_rate, 0.0f, mDiversityRateHost, mDiversityRateDevice, (int*) nullptr);
    fillBuffers(setupParams.length_penalty, 0.0f, mLengthPenaltyHost, mLengthPenaltyDevice, (int*) nullptr);
    fillBuffers(setupParams.early_stopping, 1, mEarlyStoppingHost, mEarlyStoppingDevice, (int*) nullptr);
    TLLM_LOG_TRACE("%s stop", __PRETTY_FUNCTION__);
}

__global__ void updateIndirCacheKernel(int* tgt_indir_cache, int const* src_indir_cache, BeamHypotheses bh,
    int const* input_lengths, int max_attention_window, int sink_token_length)
{
    // Update indirections from steps `input_length[bb_id]` to step `sequence_lengths[bb_id]`
    int const time_step = threadIdx.x + blockIdx.x * blockDim.x;
    int const bb_id = blockIdx.y;
    int const beam_width{bh.beam_width};
    int const batch_id = bb_id / beam_width;
    int const beam_id = bb_id % beam_width;
    int const current_step{bh.seq_len[bb_id] - 1}; // the sequence_lengths is updated, need to minus 1
    int const input_length{input_lengths == nullptr ? 0 : input_lengths[bb_id]};

    // Return early when the bb_id or timestep is out of the bound
    // No update for the indices of context part since KV Cache is shared and fixed for context part
    if (bb_id >= beam_width * bh.local_batch_size || time_step >= bh.max_seq_len || time_step < input_length
        || time_step < (bh.max_seq_len - max_attention_window) || bh.finished[bb_id].isFinished())
    {
        return;
    }
    int time_step_circ = time_step;
    if (time_step_circ >= sink_token_length)
    {
        time_step_circ
            = sink_token_length + (time_step - sink_token_length) % (max_attention_window - sink_token_length);
    }

    // for the parent_ids, we will still keep it for all past tokens (i.e. bh.max_seq_len)
    int const src_beam = bh.parent_ids_ptr[batch_id][beam_id * bh.max_seq_len + current_step];

    // for the indir tables, we have the cyclic kv cache.
    uint32_t const tgt_offset
        = batch_id * beam_width * max_attention_window + beam_id * max_attention_window + time_step_circ;
    uint32_t const src_offset
        = batch_id * beam_width * max_attention_window + src_beam * max_attention_window + time_step_circ;

    tgt_indir_cache[tgt_offset] = (time_step == current_step) ? beam_id : src_indir_cache[src_offset];
}

void updateIndirCacheKernelLauncher(int* tgt_cache_indirection, int const* src_cache_indirection, BeamHypotheses& bh,
    int const* input_lengths, int max_attention_window, int sink_token_length, hipStream_t stream)
{
    int const max_seq_len_aligned = (bh.max_seq_len + 31) / 32;
    dim3 const grid(max_seq_len_aligned, bh.local_batch_size * bh.beam_width);
    updateIndirCacheKernel<<<grid, 32, 0, stream>>>(
        tgt_cache_indirection, src_cache_indirection, bh, input_lengths, max_attention_window, sink_token_length);
}

template <typename T>
void BeamSearchLayer<T>::forward(OutputParams& op, ForwardParams const& fp)
{
    TLLM_LOG_TRACE("%s", __PRETTY_FUNCTION__);

    TLLM_CHECK_WITH_INFO(op.beamHypotheses, std::string("Output BeamHypotheses is not set."));
    TLLM_CHECK_WITH_INFO(op.sequence_length->template getPtr<int>() != nullptr || mLengthPenaltyDevice == nullptr,
        std::string("Current sequence lengths must be set for length penalty computation."));
    TLLM_CHECK_WITH_INFO(fp.ite == 0, "Pipeline Parallelism is not supported yet !");

    BeamHypotheses& bh{*op.beamHypotheses};
    bh.batch_size = static_cast<std::int32_t>(op.output_ids_ptr.shape[0]);
    bh.beam_width = static_cast<std::int32_t>(op.output_ids_ptr.shape[1]);
    bh.ite = fp.ite;
    bh.local_batch_size = fp.logits.shape[0];
    bh.max_seq_len = static_cast<std::int32_t>(op.output_ids_ptr.shape[2]);
    bh.vocab_size = mVocabSizePadded;
    bh.diversity_rates = mDiversityRateDevice;
    bh.length_penalties = mLengthPenaltyDevice;
    bh.early_stoppings = mEarlyStoppingDevice;
    // bh.input_lengths = (fp.input_lengths) ? fp.input_lengths->template getPtr<int const>() : nullptr;
    bh.end_ids = fp.end_ids.template getPtr<int const>();
    bh.log_probs = (op.output_log_probs) ? op.output_log_probs->template getPtr<float>() : nullptr;
    bh.seq_len = op.sequence_length->template getPtr<int>();
    bh.cum_log_probs = op.cum_log_probs->template getPtr<float>();
    bh.finished = reinterpret_cast<FinishedState*>(op.finished->template getPtr<FinishedState::UnderlyingType>());
    bh.output_ids_ptr = op.output_ids_ptr.template getPtr<int*>();
    bh.parent_ids_ptr = op.parent_ids_ptr.template getPtr<int*>();

    T const* logits = fp.logits.template getPtr<T>();
    T const* bias = static_cast<T const*>(nullptr);
    TLLM_CHECK_WITH_INFO(mWorkspaceSize >= 2 * bh.batch_size * bh.beam_width * bh.beam_width * 2,
        std::string("Workspace size is not enough for topk softmax."));

    invokeTopkSoftMax(logits, bias, mWorkspace, bh, mStream);
    sync_check_cuda_error();

    if (bh.beam_width > 1)
    {
        auto* const input_lengths = fp.input_lengths ? fp.input_lengths->template getPtr<int const>() : nullptr;
        auto tgt_ci = op.tgt_cache_indirection.template getPtr<int>();
        auto src_ci = fp.src_cache_indirection.template getPtr<int const>();

        updateIndirCacheKernelLauncher(
            tgt_ci, src_ci, bh, input_lengths, fp.max_attention_window, fp.sink_token_length, mStream);
        sync_check_cuda_error();
    }
}

template <typename T>
void BeamSearchLayer<T>::allocateBuffer(runtime::SizeType const batch_size, runtime::SizeType const beam_width)
{
    TLLM_LOG_TRACE("%s start", __PRETTY_FUNCTION__);
    int const nPadBeamWidth = padToNextPowerOfTwo(beam_width);
    // Unit of mWorkspaceSize is number of elements (not Byte), align to 4 for further optimization
    size_t nTopK = batch_size * nPadBeamWidth * nPadBeamWidth * 2;
    size_t nTempBuffer = batch_size * nPadBeamWidth * nMaxVocabPartForStage1FastKernel * (2 * (nPadBeamWidth * 2) + 2);
    mWorkspaceSize = roundUp(nTopK, 4) * 2 + roundUp(nTempBuffer, 4);
    mWorkspace = mAllocator->reMalloc(mWorkspace, sizeof(float) * mWorkspaceSize, true);
    mDiversityRateDevice = mAllocator->reMalloc(mDiversityRateDevice, sizeof(float) * batch_size, false);
    mLengthPenaltyDevice = mAllocator->reMalloc(mLengthPenaltyDevice, sizeof(float) * batch_size, false);
    mEarlyStoppingDevice = mAllocator->reMalloc(mEarlyStoppingDevice, sizeof(int) * batch_size, false);
    mIsAllocateBuffer = true;
    TLLM_LOG_TRACE("%s stop", __PRETTY_FUNCTION__);
}

template <typename T>
void BeamSearchLayer<T>::freeBuffer()
{
    TLLM_LOG_TRACE("%s start", __PRETTY_FUNCTION__);
    if (mIsAllocateBuffer)
    {
        mAllocator->free((void**) (&mWorkspace));
        mAllocator->free((void**) (&mDiversityRateDevice));
        mAllocator->free((void**) (&mLengthPenaltyDevice));
        mAllocator->free((void**) (&mEarlyStoppingDevice));
        mIsAllocateBuffer = false;
    }
    TLLM_LOG_TRACE("%s stop", __PRETTY_FUNCTION__);
}

template class BeamSearchLayer<float>;
template class BeamSearchLayer<half>;

} // namespace layers
} // namespace tensorrt_llm
